#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

#define M 100000
#define m 256 //threads per block

// this time global indexing
__global__ void vec_add(int *v1, int *v2, int *v3){

    __shared__ int s1[m];
    __shared__ int s2[m];

    int idx = threadIdx.x;
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;

    if(gidx < M){
        s1[idx] = v1[gidx]; //filling the shared memory 
        s2[idx] = v2[gidx];  
    }

    __syncthreads();

    if(gidx < M){
        v3[gidx] = s2[idx] + s1[idx];
    }   
}

int main(){

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int vec1[M], vec2[M], vec3[M]; //arrays allocated on the host 
    int *p1, int *p2, int *p3;  //pointers for memory assigned on the gpu

    float memAlloc_time;
    hipEventRecord(start);
    // pointers pointing to memory on gpu
    hipMalloc((void**)&p1, M * sizeof(int));
    hipMalloc((void**)&p2, M * sizeof(int));
    hipMalloc((void**)&p3, M * sizeof(int));
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&memAlloc_time, start, stop);
    printf("Memory allocation time: %.3f ms\n", memAlloc_time);
    // let's assign some value to vectors
    for (int i=0; i<M; i++){
        vec1[i] = i;
        vec2[i] = i * 2;
    }
    float H2D_time;
    hipEventRecord(start);
    //copying these vectors to gpu memory
    hipMemcpy(p1,vec1,M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(p2,vec2,M * sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&H2D_time, start, stop);
    printf("Host to Device transfer time: %.3f ms\n", H2D_time);

    // let's use multilple blocks
    // int blocks = (M + m -1)/m; //all elements should be covered
    int blocks = 8*20; //all elements should be covered
    int minGridSize, blockSize;
    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, vec_add, 0, 0);
    // printf("%d,%d\n",minGridSize,blockSize);
    // let's call the kernel 

    float kernel_time;
    hipEventRecord(start);
    vec_add<<<blocks,m>>>(p1,p2,p3);
    hipDeviceSynchronize();
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernel_time, start, stop);
    printf("Kernel execution time: %.3f ms\n", kernel_time);

    float D2H_time;
    hipEventRecord(start);
    hipMemcpy(vec3,p3,M * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&D2H_time, start, stop);
    printf("Device to Host transfer time: %.3f ms\n", D2H_time);
    
    // Calculate throughput
    float total_time = memAlloc_time + H2D_time + kernel_time + D2H_time;
    float bandwidth = (3 * M * sizeof(int)) / (total_time * 1e6); // GB/s
    printf("\nPerformance metrics:\n");
    printf("Total time: %.3f ms\n", total_time);
    printf("Effective Bandwidth: %.2f GB/s\n", bandwidth);

    printf("printing the output vector\n");

    for(int i= 0; i<5; i++){
        printf("%d + %d = %d\n", vec1[i], vec2[i], vec3[i]);
    }
    
    hipFree(p1);
    hipFree(p2);
    hipFree(p3);
    
    return 0;
}


