#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include <torch/types.h>


#define CHECK_CUDA(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    }

//rememeber inputs are 4d tensors -> (b,h,s,d)
//B_r -> total elements in each query tile
//B_c -> total elements in each key and value tile  
__global__ void flash_attn(float *K, float *V, float *Q,float *O, float *l, float *m,
    int T_r, int T_c, int b, int h, int s, int d,int B_c, int B_r, float scale){
 
    extern __shared__ float smem[];

    float *k = smem;
    float *v = k + B_c * d;
    float *q = v +  B_c * d;
    float *S = q + B_r * d;

    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;
    int x = threadIdx.x;

    //shared memory for local values
    float *local_max = S + B_r*B_c;  //B_r
    float *local_sum = local_max + B_r; //B_r
    float *row_max = local_sum + B_r;
    float *row_sum = row_max + B_r;
    float *prev_max = row_sum + B_r;
    float *prev_sum = prev_max + B_r;

    for(int j = 0 ; j < T_c; j++){

        //loading the K and V tiles
        int col = x % B_r;
        int row = x / B_r;
        for (int c = col; c < d ; c += B_r){ //coalescing
            k[row*d + c] = K[batch_idx * (h*s*d) + head_idx * (s*d) + (B_c *j*d + row*d)  + c];
            v[row*d + c] = V[batch_idx * (h*s*d) + head_idx * (s*d) + (B_c *j*d + row*d)  + c];
        }
        __syncthreads();
        
        for(int i = 0; i < T_r; i++){
            
            //loading the Q tiles
            int col = x % B_c;
            int row = x / B_c;

            for(int c = col ; c<d ; c += B_c){ //coalescing
                q[row*d + c]  = Q[batch_idx * (h*s*d) + head_idx * (s*d) + (B_r *i*d + row*d) + c];
            }
            __syncthreads();
          
            //computing the dot product
            float S_ij = 0.0f;
            for(int common = 0 ; common < d; common++){
                S_ij +=  q[row*d + common] * k[col*d + common];
            }
            S[row * B_c + col] = scale * S_ij;

            //computing the local max and local sum
            if(col==0){
              float local_mij = -INFINITY;
              float local_lij = 0.0f;
              for (int common = 0; common <B_c; common++){
                float curr_value = S[row * B_c + common];
                if(curr_value > local_mij){
                  local_lij = local_lij * expf(local_mij - curr_value);
                  local_mij = curr_value;
                }
                local_lij += expf(curr_value - local_mij);
              }
            local_max[row] = local_mij;
            local_sum[row] = local_lij;

            prev_max[row] = m[batch_idx * (h*s) + head_idx * s + B_r * i + row];
            prev_sum[row] = l[batch_idx * (h*s) + head_idx * s + B_r * i + row]; 
            row_max[row] = max(prev_max[row],local_max[row]);
            row_sum[row] = expf(prev_max[row]- row_max[row]) * prev_sum[row] + expf(local_max[row] - row_max[row]) * local_sum[row];
            }
            __syncthreads();

            
            //computing the final output
            for(int c = col; c < d; c += B_c){
              float output_sum = 0.0f;
              for(int common = 0; common < B_c; common++){

                output_sum += expf(S[row * B_c + common] - row_max[row]) * v[common*d+ c];
              }
              int idx = batch_idx * (h*s*d) + head_idx * (s*d) + (B_r *i*d + row*d) + c;
              O[idx] = output_sum /row_sum[row] + (O[idx] * expf(prev_max[row] - row_max[row]) * prev_sum[row]) / row_sum[row];
            }
            
            if(col==0){
              l[batch_idx * (h*s) + head_idx * s + B_r * i + row] = row_sum[row];
              m[batch_idx * (h*s) + head_idx * s + B_r * i + row] = row_max[row];
            }
            __syncthreads();
        }  
    }
}

torch::Tensor fa_forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
  const int Bc = 16;
  const int Br = 16;

  int B = Q.size(0);
  int nh = Q.size(1);
  int N = Q.size(2);
  int d = Q.size(3);

  int Tc = ceil((float)N / Bc);
  int Tr = ceil((float)N / Br);
  float scale = 1.0 / sqrt(d);

  auto O = torch::zeros_like(Q);
  auto l = torch::zeros({B,nh,N});
  auto m = torch::full({B,nh,N} ,-INFINITY);
  torch::Device device(torch::kCUDA);
  l = l.to(device);
  m = m.to(device);

  const int smem_size = (2 * Bc * d + Br * d + Br * Bc + 6*Br) * sizeof(float);
  int max_sram_size;
  hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
  printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, smem_size);

  dim3 grid_size(B, nh);     
  dim3 block_size(Br*Bc);  

  flash_attn<<<grid_size, block_size, smem_size>>>(
    K.data_ptr<float>(), V.data_ptr<float>(), Q.data_ptr<float>(), O.data_ptr<float>(), l.data_ptr<float>(), m.data_ptr<float>(),
    Tr, Tc, B, nh, N, d, Bc, Br,scale);
  return O;
}

int main() {
    int b = 1; // Batch size
    int h = 1; // Number of heads
    int s = 8; // Sequence length
    int d = 8; // Embedding dimension
    int B_c = 4; // Block size for keys/values
    int B_r = 4; // Block size for queries
    int T_r = s / B_r;
    int T_c = s / B_c;

    float scale = 1/sqrtf(d);
    size_t size = b * h * s * d * sizeof(float);
    float *h_K = (float*)malloc(size);
    float *h_V = (float*)malloc(size);
    float *h_Q = (float*)malloc(size);
    float *h_O = (float*)malloc(size);
    float *h_l = (float*)malloc(s * sizeof(float));
    float *h_m = (float*)malloc(s * sizeof(float));

    for (int i = 0; i < b * h * s * d; i++) {
        h_K[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
        h_V[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
        h_Q[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
        h_O[i] = 0.0f;
    }
    for (int i = 0; i < s; i++) {
        h_l[i] = 1.0f;
        h_m[i] = -INFINITY;
    }

    float *d_K, *d_V, *d_Q, *d_O, *d_l, *d_m;
    CHECK_CUDA(hipMalloc((void**)&d_K, size));
    CHECK_CUDA(hipMalloc((void**)&d_V, size));
    CHECK_CUDA(hipMalloc((void**)&d_Q, size));
    CHECK_CUDA(hipMalloc((void**)&d_O, size));
    CHECK_CUDA(hipMalloc((void**)&d_l, s * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_m, s * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_K, h_K, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_V, h_V, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Q, h_Q, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_O, h_O, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_l, h_l, s * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_m, h_m, s * sizeof(float), hipMemcpyHostToDevice));

    dim3 gridDim(b, h);
    dim3 blockDim(B_r, B_c);
    size_t sharedMemSize = (2 * B_c * d + B_r * d + B_r * B_c) * sizeof(float);

    flash_attn<<<gridDim, blockDim, sharedMemSize>>>(d_K, d_V, d_Q, d_O, d_l, d_m,
                                                      T_r, T_c, b, h, s, d, B_c, B_r,scale);
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(h_O, d_O, size, hipMemcpyDeviceToHost));

    printf("Output tensor O (partial view):\n");
    for (int i = 0; i < s; i++) {
        for (int j = 0; j < d; j++) {
            printf("%.3f ", h_O[i * d + j]);
        }
        printf("\n");
    }

    free(h_K);
    free(h_V);
    free(h_Q);
    free(h_O);
    free(h_l);
    free(h_m);
    CHECK_CUDA(hipFree(d_K));
    CHECK_CUDA(hipFree(d_V));
    CHECK_CUDA(hipFree(d_Q));
    CHECK_CUDA(hipFree(d_O));
    CHECK_CUDA(hipFree(d_l));
    CHECK_CUDA(hipFree(d_m));

    return 0;
}

