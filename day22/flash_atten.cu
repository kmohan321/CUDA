#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include <torch/types.h>


#define CHECK_CUDA(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            printf("CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    }

//rememeber inputs are 4d tensors -> (b,h,s,d)
//B_r -> total elements in each query tile
//B_c -> total elements in each key and value tile  
__global__ void flash_attn(float *K, float *V, float *Q,float *O, float *l, float *m,
    int T_r, int T_c, int b, int h, int s, int d,int B_c, int B_r, float scale){
 
    extern __shared__ float smem[];

    float *k = smem;
    float *v = k + B_c * d;
    float *q = v +  B_c * d;
    float *S = q + B_r * d;

    int batch_idx = blockIdx.x;
    int head_idx = blockIdx.y;
    int x = threadIdx.x;

    
    for(int j = 0 ; j < T_c; j++){

        //loading the K and V tiles
        for (int c = 0; c < d ; c++){
            k[x*d + c] = K[batch_idx * (h*s*d) + head_idx * (s*d) + (B_c *j*d + x*d) + c];
            v[x*d + c] = V[batch_idx * (h*s*d) + head_idx * (s*d) + (B_c *j*d + x*d) + c];
        }
        __syncthreads();
        
        for(int i = 0; i < T_r; i++){
            
            //loading the Q tiles
            for(int c = 0; c<d ; c++){
                q[x*d + c]  = Q[batch_idx * (h*s*d) + head_idx * (s*d) + (B_r *i*d + x*d) + c];
            }
            __syncthreads();
            
            //local values for each query 
            float l_i = l[batch_idx * (h*s) + head_idx * s + B_r * i + x]; 
            float m_i = m[batch_idx * (h*s) + head_idx * s + B_r * i + x];

            //computing the dot product and max localvalue
            float local_mij = -INFINITY;
            for(int l_row = 0; l_row < B_c; l_row++){
                float S_ij = 0.0f;
                for(int common = 0 ; common < d; common++){
                    S_ij +=  q[x*d + common] * k[l_row*d + common];
                }
                S[x * B_c + l_row] = scale * S_ij;
                local_mij = fmax(local_mij,S_ij);
            }
            __syncthreads();

            //local sum calculation
            float local_lij = 0.0f;
            for(int l_row = 0; l_row < B_c; l_row++){
                float curr_value = S[ x* B_c + l_row];
                local_lij += __expf(curr_value-local_mij);
            }

            float m_i_ = fmax(m_i,local_mij);
            float l_i_ = __expf(m_i - m_i_) * l_i + __expf(local_mij - m_i_) * local_lij;
            
            //computing the final output
            for(int l_row = 0; l_row < d; l_row ++){
                float local_sum = 0.0f;
                for(int common = 0; common < B_c; common++){
                    local_sum += __expf(S[x * B_c + common]-m_i_) * v [common*d + l_row];
                }
                int idx = batch_idx * (h*s*d) + head_idx * (s*d) + (B_r *i*d + x*d) + l_row;
                O[idx] = local_sum /l_i_ + (O[idx] * __expf(m_i - m_i_) * l_i ) / l_i_;
            }
        
            l[batch_idx * (h*s) + head_idx * s + B_r * i + x] = l_i_;
            m[batch_idx * (h*s) + head_idx * s + B_r * i + x] = m_i_;
            __syncthreads();
        }  
    }
}


torch::Tensor fa_forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
  const int Bc = 32;
  const int Br = 32;

  int B = Q.size(0);
  int nh = Q.size(1);
  int N = Q.size(2);
  int d = Q.size(3);

  int Tc = ceil((float)N / Bc);
  int Tr = ceil((float)N / Br);
  float scale = 1.0 / sqrt(d);

  auto O = torch::zeros_like(Q);
  auto l = torch::zeros({B,nh,N});
  auto m = torch::full({B,nh,N} ,-INFINITY);
  torch::Device device(torch::kCUDA);
  l = l.to(device);
  m = m.to(device);

  const int smem_size = (2 * Bc * d + Br * d + Br * Bc) * sizeof(float);
  int max_sram_size;
  hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
  printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, smem_size);

  dim3 grid_size(B, nh);     
  dim3 block_size(Br);  

  flash_attn<<<grid_size, block_size, smem_size>>>(
    K.data_ptr<float>(), V.data_ptr<float>(), Q.data_ptr<float>(), O.data_ptr<float>(), l.data_ptr<float>(), m.data_ptr<float>(),
    Tr, Tc, B, nh, N, d, Bc, Br,scale);
  return O;
}

int main() {
    int b = 1; // Batch size
    int h = 1; // Number of heads
    int s = 8; // Sequence length
    int d = 8; // Embedding dimension
    int B_c = 4; // Block size for keys/values
    int B_r = 4; // Block size for queries
    int T_r = s / B_r;
    int T_c = s / B_c;

    float scale = 1/sqrtf(d);
    size_t size = b * h * s * d * sizeof(float);
    float *h_K = (float*)malloc(size);
    float *h_V = (float*)malloc(size);
    float *h_Q = (float*)malloc(size);
    float *h_O = (float*)malloc(size);
    float *h_l = (float*)malloc(s * sizeof(float));
    float *h_m = (float*)malloc(s * sizeof(float));

    for (int i = 0; i < b * h * s * d; i++) {
        h_K[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
        h_V[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
        h_Q[i] = ((float)rand() / RAND_MAX) * 2.0 - 1.0;
        h_O[i] = 0.0f;
    }
    for (int i = 0; i < s; i++) {
        h_l[i] = 1.0f;
        h_m[i] = -INFINITY;
    }

    float *d_K, *d_V, *d_Q, *d_O, *d_l, *d_m;
    CHECK_CUDA(hipMalloc((void**)&d_K, size));
    CHECK_CUDA(hipMalloc((void**)&d_V, size));
    CHECK_CUDA(hipMalloc((void**)&d_Q, size));
    CHECK_CUDA(hipMalloc((void**)&d_O, size));
    CHECK_CUDA(hipMalloc((void**)&d_l, s * sizeof(float)));
    CHECK_CUDA(hipMalloc((void**)&d_m, s * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_K, h_K, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_V, h_V, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_Q, h_Q, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_O, h_O, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_l, h_l, s * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_m, h_m, s * sizeof(float), hipMemcpyHostToDevice));

    dim3 gridDim(b, h);
    dim3 blockDim(B_r, B_c);
    size_t sharedMemSize = (2 * B_c * d + B_r * d + B_r * B_c) * sizeof(float);

    flash_attn<<<gridDim, blockDim, sharedMemSize>>>(d_K, d_V, d_Q, d_O, d_l, d_m,
                                                      T_r, T_c, b, h, s, d, B_c, B_r,scale);
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(h_O, d_O, size, hipMemcpyDeviceToHost));

    printf("Output tensor O (partial view):\n");
    for (int i = 0; i < s; i++) {
        for (int j = 0; j < d; j++) {
            printf("%.3f ", h_O[i * d + j]);
        }
        printf("\n");
    }

    free(h_K);
    free(h_V);
    free(h_Q);
    free(h_O);
    free(h_l);
    free(h_m);
    CHECK_CUDA(hipFree(d_K));
    CHECK_CUDA(hipFree(d_V));
    CHECK_CUDA(hipFree(d_Q));
    CHECK_CUDA(hipFree(d_O));
    CHECK_CUDA(hipFree(d_l));
    CHECK_CUDA(hipFree(d_m));

    return 0;
}

