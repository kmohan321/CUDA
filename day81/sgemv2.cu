#include<stdio.h>
#include<hip/hip_runtime.h>

/*
consider a this situation what's happening is this
-> thread 0-> access 0 th memory location
-> thread 1 -> access (0 + N) th memory location and so on ...
which is uncoalesced access
- next -> this is the coalesced kernel
*/
template <int M, int N>
__global__ void sgemv_3(float *matrix, float *vector, float *ouput){

    int row = blockIdx.x;
    int idx = threadIdx.x;
    int offset = row * N;

    if(row >= M) return;
    extern __shared__ float smem[];

    float local_sum = 0.0f;
    for(int i = idx ; i < N; i += blockDim.x){
      local_sum += matrix[offset + i] * vector[i];
    }

    smem[idx] = local_sum;
    __syncthreads();

    for(int i = blockDim.x /2 ; i>0; i/=2){
      if(idx<i){
        smem[idx] += smem[idx + i]; 
      }
      __syncthreads();
    }

    if(idx==0){
      ouput[row] = smem[0];
    }
}


float compute_gflops(int M, int N, float ms) {
  return (2 * M * N) / (ms * 1e6);
}

float compute_peak_gflops(float gflops, float THEORETICAL_MAX_GFLOPS) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  return (gflops / THEORETICAL_MAX_GFLOPS) * 100;
}

float compute_peak_memory_bandwidth(int M, int N, float ms, float THEORETICAL_MAX_MEMORY_BANDWIDTH) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  size_t totalFloats = (size_t)(M * N + N + M);
  float totalBytes = (float)totalFloats * sizeof(float);

  float secs = ms / 1000.0f;
  float gbPerSec = (totalBytes / secs) / 1.0e9;

  return (gbPerSec / THEORETICAL_MAX_MEMORY_BANDWIDTH) * 100;
}

void print_kernel_essentials(int M, int N, float ms, float THEORETICAL_MAX_GFLOPS, float THEORETICAL_MAX_MEMORY_BANDWIDTH) {
  float gflops = compute_gflops(M, N, ms);
  printf(">> Execution time: %f ms\n", ms);
  printf(">> Achieved (GFLOPS): %f\n", gflops);
  printf(">> Theoretical max (GFLOPS): %f\n", THEORETICAL_MAX_GFLOPS);
  printf(">> Maximum memory bandwidth: %f GB/s\n", THEORETICAL_MAX_MEMORY_BANDWIDTH);
  printf(">> Achieves %f %% of peak GFLOPS\n", compute_peak_gflops(gflops, THEORETICAL_MAX_GFLOPS));
  printf(">> Achieves %f %% of peak Memory Bandwidth\n", compute_peak_memory_bandwidth(M, N, ms, THEORETICAL_MAX_MEMORY_BANDWIDTH));
}


void fill_matrix(float *matrix, int M ,int N){
  for(int i = 0; i < M*N ; i++){
    matrix[i] = rand() / RAND_MAX;
  }
}

int main() {
  const int M = 4096;
  const int N = 4096;
  const int tile = 256;

  // float h_matrix[M * N] = {
  //     1, 2, 3,
  //     4, 5, 6,
  //     7, 8, 9,
  //     10, 11, 12
  // };
  // float h_vector[N] = {1, 1, 1};
  // float h_output[M] = {0};

  float *h_matrix = (float*)malloc(M*N*sizeof(float));
  float *h_vector = (float*)malloc(N*sizeof(float));
  float *h_output = (float*)malloc(M * sizeof(float));

  fill_matrix(h_matrix,M,N);
  fill_matrix(h_vector,1,N);

  float *d_matrix, *d_vector, *d_output;
  hipMalloc(&d_matrix, M * N * sizeof(float));
  hipMalloc(&d_vector, N * sizeof(float));
  hipMalloc(&d_output, M * sizeof(float));

  hipMemcpy(d_matrix, h_matrix, M * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_vector, h_vector, N * sizeof(float), hipMemcpyHostToDevice);

  dim3 blocksize(tile);
  dim3 grid(M);
  int smem_size = tile * sizeof(float);
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);
  sgemv_3<M, N><<<grid, blocksize, smem_size>>>(d_matrix, d_vector, d_output);
  hipEventRecord(stop);

  hipMemcpy(h_output, d_output, M * sizeof(float), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  float ms = 0.0f;
  hipEventElapsedTime(&ms, start, stop);

  // printf("Result (Matrix * Vector):\n");
  // for (int i = 0; i < M; ++i) {
  //     printf("%.f\n", h_output[i]);
  // }

  // Theoretical values (adjust for your GPU)
  float THEORETICAL_MAX_GFLOPS = 10000.0f;              // Example: NVIDIA A100 ~19.5 TFLOPS (FP32)
  float THEORETICAL_MAX_MEMORY_BANDWIDTH = 1555.0f;     // GB/s (adjust as per GPU, ex: A100 HBM2)

  print_kernel_essentials(M, N, ms, THEORETICAL_MAX_GFLOPS, THEORETICAL_MAX_MEMORY_BANDWIDTH);

  hipFree(d_matrix);
  hipFree(d_vector);
  hipFree(d_output);

  return 0;
}



